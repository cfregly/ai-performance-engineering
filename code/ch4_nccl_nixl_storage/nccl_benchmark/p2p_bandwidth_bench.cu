// p2p_bandwidth_bench.cu
// Measures GPU peer-to-peer bandwidth between device 0 and 1.
// Requires: CUDA 13.0, C++17

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdio>

#define CHECK(call)                                                         \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::fprintf(stderr, "CUDA error %s:%d: %s\n",                \
                         __FILE__, __LINE__, hipGetErrorString(err));      \
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    } while(0)

int main() {
    int devCount = 0;
    CHECK(hipGetDeviceCount(&devCount));
    if (devCount < 2) {
        std::cerr << "Need at least 2 GPUs for peer-to-peer benchmark\n";
        return 1;
    }

    // Enable peer access
    CHECK(hipSetDevice(0));
    CHECK(hipDeviceEnablePeerAccess(1, 0));
    CHECK(hipSetDevice(1));
    CHECK(hipDeviceEnablePeerAccess(0, 0));

    std::vector<size_t> sizes = {1<<20, 4<<20, 16<<20, 64<<20, 256<<20, 1<<30}; //bytes
    std::cout << "Size(MB)    Bandwidth(GB/s)\n";

    for (auto bytes : sizes) {
        // allocate on src and dst
        CHECK(hipSetDevice(0));
        void* src = nullptr;
        CHECK(hipMalloc(&src, bytes));
        CHECK(hipMemset(src, 0, bytes));

        CHECK(hipSetDevice(1));
        void* dst = nullptr;
        CHECK(hipMalloc(&dst, bytes));

        // events
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));

        // record, copy, record
        CHECK(hipEventRecord(start, 0));
        CHECK(hipMemcpyPeer(dst, 1, src, 0, bytes));
        CHECK(hipEventRecord(stop, 0));
        CHECK(hipEventSynchronize(stop));

        float ms = 0.0f;
        CHECK(hipEventElapsedTime(&ms, start, stop));

        float gb = bytes / 1e9f;
        float bw = gb / (ms / 1e3f);
        std::printf("%8zu    %10.2f\n", bytes/(1<<20), bw);

        // cleanup
        hipEventDestroy(start);
        hipEventDestroy(stop);
        CHECK(hipFree(src));
        CHECK(hipFree(dst));
    }

    return 0;
}
