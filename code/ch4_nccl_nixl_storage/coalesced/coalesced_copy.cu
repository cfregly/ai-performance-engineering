// coalesced_copy.cu
// Demo: Coalesced Global Memory Copy
// Hardware: Grace-Blackwell (sm_90) or Hopper (sm_80)
// CUDA: 13.0  C++17
// Build: make

#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

#define CHECK_CUDA(call) do {                                   \
    hipError_t err = call;                                     \
    if (err != hipSuccess) {                                   \
        std::cerr << "CUDA Error: " << hipGetErrorString(err); \
        std::exit(EXIT_FAILURE);                                \
    } } while(0)

// Coalesced copy kernel: each thread copies one float
__global__ void coalescedCopy(const float* __restrict__ in,
                              float* __restrict__ out,
                              int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

int main() {
    // Problem size
    const int N = 1 << 24;              // 16M elements (~64 MB per buffer)
    const size_t bytes = N * sizeof(float);

    // Allocate host memory
    float *h_in  = nullptr, *h_out = nullptr;
    h_in  = (float*)malloc(bytes);
    h_out = (float*)malloc(bytes);
    assert(h_in && h_out);

    // Initialize input
    for (int i = 0; i < N; ++i) {
        h_in[i] = static_cast<float>(i);
    }

    // Device pointers
    float *d_in = nullptr, *d_out = nullptr;
    CHECK_CUDA(hipMalloc(&d_in,  bytes));
    CHECK_CUDA(hipMalloc(&d_out, bytes));

    // Copy input to GPU
    CHECK_CUDA(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    // Record time with CUDA events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));
    coalescedCopy<<<grid, block>>>(d_in, d_out, N);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));
    std::cout << "Kernel time: " << ms << " ms\n";

    // Copy back and verify
    CHECK_CUDA(hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost));
    for (int i = 0; i < N; i += (N/4)) {
        assert(h_out[i] == h_in[i]);
    }

    std::cout << "Result verified!\n";

    // Cleanup
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
